﻿
#include "hip/hip_runtime.h"
#include ""
#include "cuda_aes256.cuh"
#include <iostream>

using namespace std;
using namespace cuda_aes;
using namespace cuda_aes::file;

int main() {
	std::string txt = "test.txt";
	CUDA_AES_FileReader reader(txt, 10, 10);
	reader.start();
	return 0;
}
